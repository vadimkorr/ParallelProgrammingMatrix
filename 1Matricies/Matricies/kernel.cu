#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <conio.h>
#include <stdlib.h>

// Thread block size
#define BLOCK_SIZE 64 //submatrix size 
#define N 4096 //matrix size is N*N 
#define MULTIPLIER_A 5
#define MULTIPLIER_B 8
__global__ void matMatMult(int* a, int* b, int n, int* c);
__global__ void matMultByConst(int* a, int k, int n, int* c);
__global__ void sumMat(int* a, int* b, int n, int* c);//sum A, B, C, result to C

void PrintMatrix(int* a, int n);
void InitMatrix(int* a, int n, int initVal);

int main (int argc, char* argv[])
{
	int numBytes = N * N * sizeof(int);
	
	//allocate host memory 
	int* a = new int[N*N];
	int* b = new int[N*N];
	int* c = new int[N*N];

	//init matricies
	InitMatrix(a, N, 1);
	InitMatrix(b, N, 1);

	//allocate device memory 
	int* adev = NULL;
	int* bdev = NULL;
	int* cdev = NULL;
	hipMalloc((void**)&adev, numBytes);
	hipMalloc((void**)&bdev, numBytes);
	hipMalloc((void**)&cdev, numBytes);

	//set kernel launch configuration 
	dim3 threads (BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks (N / threads.x, N / threads.y);

	//create cuda event handles 
	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	hipEventCreate (&start);
	hipEventCreate (&stop);

	//asynchronously issue work to the GPU (all to stream 0) 
	hipEventRecord(start, 0);
	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);
	
	matMatMult<<<blocks, threads>>>(adev, bdev, N, cdev); hipDeviceSynchronize();
	matMultByConst << <blocks, threads >> >(adev, MULTIPLIER_A, N, adev); hipDeviceSynchronize();
	matMultByConst << <blocks, threads >> >(bdev, MULTIPLIER_B, N, bdev); hipDeviceSynchronize();
	sumMat << <blocks, threads >> >(adev, bdev, N, cdev); hipDeviceSynchronize();

	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);
	PrintMatrix(c, N);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	
	//print the cpu and gpu times 
	printf("time spent executing by the GPU: %.2f ms\n", gpuTime);

	//release resources 
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(adev);
	hipFree(bdev);
	hipFree(cdev);
	delete a;
	delete b;
	delete c;
	getch();
	return 0;
}

__global__ void matMatMult(int* a, int* b, int n, int* c)
{
	int bx = blockIdx.x;//block index
	int by = blockIdx.y;
	int tx = threadIdx.x;//thread index inside block
	int ty = threadIdx.y;
	int ia = n * (BLOCK_SIZE * by + ty);//offset for a[i][0]	
	int ib = BLOCK_SIZE * bx + tx;//offset for b[0][j]
	int sum = 0;//computed subelement 

	for (int k = 0; k < n; k++)//mult row by column
	{
		sum += a[ia + k] * b[ib + k*n];
	}
	int ic = n*BLOCK_SIZE*by + BLOCK_SIZE*bx;//offset for result
	c[ic + n*ty + tx] = sum;
}

__global__ void matMultByConst(int* a, int k, int n, int* c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < N && j < N) {
		c[i*n+ j] = a[i*n + j] * k;
	}
}

__global__ void sumMat(int* a, int* b, int n, int* c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < N && j < N) {
		c[i*n + j] = a[i*n + j] + b[i*n + j] + c[i*n + j];
	}
}

/* UTILS */

void InitMatrix(int* a, int n, int initVal)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			int k = n * i + j;
			a[k] = initVal;
		}
	}
}

void PrintMatrix(int* a, int n)
{
	int rowStep = (n - 1) / 2;
	int colStep = (n - 1) / 2;

	for (int i = 0; i < n; i += rowStep)
	{
		for (int j = 0; j < n; j += colStep)
		{
			int k = n * i + j;
			printf("[%d,%d]: %d\n", i, j, a[k]);
		}
	}
	printf("\n");
}


/*

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <conio.h>
#include <stdlib.h>


// Thread block size
#define BLOCK_SIZE 5 //submatrix size 
#define N 5 //matrix size is N*N 
__global__ void matMult(int* a, int* b, int n, int* c);
void PrintMatrix(int* a, int n);
void InitMatrix(int* a, int n, int initVal);


int main (int argc, char* argv[])
{
int numBytes = N * N * sizeof(int);

//allocate host memory 
int* a = new int[N*N];
int* b = new int[N*N];
int* c = new int[N*N];

//init matricies
InitMatrix(a, N, 1);
InitMatrix(b, N, 1);

//allocate device memory 
int* adev = NULL;
int* bdev = NULL;
int* cdev = NULL;
hipMalloc((void**)&adev, numBytes);
hipMalloc((void**)&bdev, numBytes);
hipMalloc((void**)&cdev, numBytes);

//set kernel launch configuration 
dim3 threads (BLOCK_SIZE, BLOCK_SIZE);
dim3 blocks (N / threads.x, N / threads.y);

//create cuda event handles 
hipEvent_t start, stop;
float gpuTime = 0.0f;
hipEventCreate (&start);
hipEventCreate (&stop);

//asynchronously issue work to the GPU (all to stream 0) 
hipEventRecord(start, 0);
hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

matMult<<<blocks, threads>>>(adev, bdev, N, cdev);

hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);
PrintMatrix(c, N);

hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&gpuTime, start, stop);

//print the cpu and gpu times 
printf("time spent executing by the GPU: %.2f ms\n", gpuTime);

//release resources 
hipEventDestroy(start);
hipEventDestroy(stop);
hipFree(adev);
hipFree(bdev);
hipFree(cdev);
delete a;
delete b;
delete c;
getch();
return 0;
}

__global__ void matMult(int* a, int* b, int n, int* c)
{
int bx = blockIdx.x; //block index 
int by = blockIdx.y;
int tx = threadIdx.x; //thread index 
int ty = threadIdx.y;
int sum = 0.0f; //computed subelement 
int ia = n * BLOCK_SIZE * by + n * ty; //a [i][0] 
int ib = BLOCK_SIZE * bx + tx;

//C = A*B 
for (int k = 0; k < n; k++)
{
sum += a[ia + k] * b[ib + k*n];
}

//Write the block sub‐matrix to global memory; each thread writes one element 
int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
c[ic + n * ty + tx] = sum;
__syncthreads();


//A = A*5, B = B*8 
for (int k = 0; k < n; k++)
{
a[ia + k] *= 2;
b[ib + k*n] *= 3;
}
__syncthreads();

//C = A - B + C
sum = 0.0f;
ic = ia;
for (int k = 0; k < n; k++)
{
sum = a[ia + k] - b[ib + k*n] + c[ic + k];
}

// Write the block sub‐matrix to global memory; each thread writes one element 
ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
c[ic + n * ty + tx] = sum;
__syncthreads();
}


//UTILS 

void InitMatrix(int* a, int n, int initVal)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			int k = n * i + j;
			a[k] = initVal;
		}
	}
}

void PrintMatrix(int* a, int n)
{
	int rowStep = (n - 1) / 2;
	int colStep = (n - 1) / 2;

	for (int i = 0; i < n; i += rowStep)
	{
		for (int j = 0; j < n; j += colStep)
		{
			int k = n * i + j;
			printf("[%d,%d]: %d\n", i, j, a[k]);
		}
	}
	printf("\n");
}
*/